#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <unistd.h>

#define TASK_NB 8
#define WARP_SIZE 32
#define LIST_SIZE 4 //实际容量要减1
#define NEXT_TASK(ID) ((ID + 1) % LIST_SIZE)

struct Task
{
    int taskId;
    bool isFin;
    int task[WARP_SIZE];
    void set(int value)
    {
        taskId = value;
        isFin = false;
        for (int i = 0; i < WARP_SIZE; i++)
        {
            task[i] = value;
        }
    }
};

/*主机端内存*/
static struct Task *list;
static struct Task *result;
static int *flag;

/*设备端内存*/
static struct Task *devList;
static int *devFlag;

/*设备端访问主机端pinned内存*/
static struct Task *hostList;
static struct Task *hostResult;
static int *hostFlag;

/*主机端->设备端内存拷贝流*/
static hipStream_t streamHd;

/*内核执行流*/
static hipStream_t streamKernel;

/*主机端生产者*/
void *cpu_producer(void *argc)
{
    for (int i = 1; i <= TASK_NB; i++)
    {
        bool temp = false;
        while (flag[0] == NEXT_TASK(flag[1]))
        {
            if (temp == false)
            {
                printf("[cpu] 队列是满的\n");
                temp = true;
            }
        }
        int id = flag[1];
        while (list[id].isFin == true)
        {
        }
        list[id].set(i);
        flag[1] = NEXT_TASK(flag[1]);
        hipMemcpyAsync(devList + id, list + id, sizeof(struct Task), hipMemcpyHostToDevice, streamHd);
        hipMemcpyAsync(devFlag + 1, flag + 1, sizeof(int), hipMemcpyHostToDevice, streamHd);
        printf("[cpu] 在%d处插入任务%d\n", id, i);
    }
    return NULL;
}

/*设备端消费者*/
__global__ void gpu_consumer(struct Task *devList, int *devFlag, struct Task *hostList, int *hostFlag, struct Task *hostResult)
{
    int threadId = threadIdx.x;
    while (1)
    {
        __syncthreads();
        bool temp = false;
        while (devFlag[0] == devFlag[1])
        {
            if (threadId == 0)
            {
                if (temp == false)
                {
                    printf("[gpu] 队列是空的\n");
                    temp = true;
                }
            }
        }
        int id = devFlag[0];
        int task = devList[id].task[threadId];
        hostResult[id].task[threadId] = pow(task, 2);
        __syncthreads();
        if (threadId == 0)
        {
            printf("[gpu] %d处的任务%d处理完成\n", id, devList[id].taskId);
            hostList[id].isFin = true;
            devFlag[0] = NEXT_TASK(devFlag[0]);
            hostFlag[0] = devFlag[0];
        }
    }
}

/*保存结果到文件*/
void *cpu_saver(void *argc)
{
    remove("./result.txt");
    FILE *fp = fopen("./result.txt", "a+");
    fprintf(fp, "%d\t", list[0].taskId);
    while (1)
    {
        for (int id = 0; id < LIST_SIZE; id++)
        {
            if (list[id].isFin == true)
            {
                printf("%d ", list[id].taskId);
                fprintf(fp, "%d\t", list[id].taskId);
                for (int i = 0; i < WARP_SIZE; i++)
                {
                    fprintf(fp, "%d", result[id].task[i]);
                    if (i < WARP_SIZE - 1)
                    {
                        fprintf(fp, " ");
                    }
                }
                printf("%d\n", result[id].task[0]);
                fprintf(fp, "\n");
                list[id].isFin = false;
            }
        }
    }
    fclose(fp);
    return NULL;
}

/*初始化*/
void init()
{
    int listBytes = LIST_SIZE * sizeof(struct Task);
    int flagBytes = 2 * sizeof(int);
    int resultBytes = LIST_SIZE * sizeof(struct Task);

    hipHostMalloc((void **)&list, listBytes, hipHostMallocMapped, hipHostMallocDefault);
    hipHostMalloc((void **)&flag, flagBytes, hipHostMallocMapped, hipHostMallocDefault);
    hipHostMalloc((void **)&result, resultBytes, hipHostMallocMapped, hipHostMallocDefault);

    for (int i = 0; i < LIST_SIZE; i++)
    {
        list[i].set(0);
        result[i].set(0);
    }
    memset(flag, 0, flagBytes);

    hipMalloc((void **)&devList, listBytes);
    hipMalloc((void **)&devFlag, flagBytes);

    hipStreamCreate(&streamHd);
    hipStreamCreate(&streamKernel);

    hipMemcpyAsync(devList, list, listBytes, hipMemcpyHostToDevice, streamHd);
    hipMemcpyAsync(devFlag, flag, flagBytes, hipMemcpyHostToDevice, streamHd);

    hipHostGetDevicePointer<struct Task>(&hostList, (void *)list, 0);
    hipHostGetDevicePointer<int>(&hostFlag, (void *)flag, 0);
    hipHostGetDevicePointer<struct Task>(&hostResult, (void *)result, 0);
}

/*清理*/
void free()
{
    hipStreamDestroy(streamHd);
    hipStreamDestroy(streamKernel);
    hipFree(devList);
    hipFree(devFlag);
    hipHostFree(list);
    hipHostFree(flag);
    hipHostFree(result);
}

int main()
{
    init();

    pthread_t cpu_pro, cpu_sav;
    pthread_create(&cpu_sav, NULL, cpu_saver, NULL);
    gpu_consumer<<<1, WARP_SIZE, 0, streamKernel>>>(devList, devFlag, hostList, hostFlag, hostResult);
    pthread_create(&cpu_pro, NULL, cpu_producer, NULL);

    pthread_join(cpu_pro, NULL);
    hipDeviceSynchronize();
    pthread_join(cpu_sav, NULL);

    free();
    return 0;
}
/*
*vscode的工作目录必须为cuda*

rm -rf cpu_gpu_syn cpu_gpu_syn.o result.txt
/usr/local/cuda/bin/nvcc -ccbin g++ -I /usr/local/cuda/include -I /usr/local/cuda/samples/common/inc -m64 -g -G -gencode arch=compute_75,code=sm_75 -gencode arch=compute_75,code=compute_75 -o cpu_gpu_syn.o -c cpu_gpu_syn.cu
/usr/local/cuda/bin/nvcc -ccbin g++ -m64 -g -G -gencode arch=compute_75,code=sm_75 -gencode arch=compute_75,code=compute_75 -o cpu_gpu_syn cpu_gpu_syn.o -L /usr/local/cuda/lib64 -L /usr/local/cuda/samples/common/lib

./cpu_gpu_syn

cuda-gdb
file cpu_gpu_syn
r
q

rm -rf cpu_gpu_syn cpu_gpu_syn.o result.txt
*/