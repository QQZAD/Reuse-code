#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <unistd.h>

#define TASK_NB 100
#define WARP_SIZE 32
#define LIST_SIZE 6 //实际容量要减1
#define NEXT_TASK(ID) ((ID + 1) % LIST_SIZE)

struct Task
{
    int id;
    int nb;
    int *pData;
    int *pDevResult;
    int *pHostResult;
    bool isSave;
    Task()
    {
        id = 0;
        nb = 0;
        pData = NULL;
        pDevResult = NULL;
        pHostResult = NULL;
        isSave = false;
    }
};

/*主机端内存*/
static struct Task *list;
static int *flag;
static int *finTaksNb;

/*设备端访问主机端pinned内存*/
static struct Task *devList;
static int *devFlag;
static int *devFinTaksNb;

/*主机端->设备端内存拷贝流*/
static hipStream_t streamHd;

/*设备端->主机端内存拷贝流*/
static hipStream_t streamDh;

/*内核执行流*/
static hipStream_t streamKernel;

/*CUDA异常处理*/
static hipError_t err;

/*主机端生产者*/
void *cpuProducer(void *argc)
{
    for (int i = 1; i <= TASK_NB; i++)
    {
        bool temp = false;
        while (flag[0] == NEXT_TASK(flag[1]))
        {
            if (temp == false)
            {
                printf("[cpu] 队列是满的\n");
                temp = true;
            }
        }
        int cur = flag[1];
        list[cur].nb = rand() % (WARP_SIZE - 1 + 1) + 1;
        int bytes = sizeof(int) * list[cur].nb;
        int *data = (int *)malloc(bytes);
        for (int j = 0; j < list[cur].nb; j++)
        {
            data[j] = i;
        }
        err = hipMemcpyAsync(list[cur].pData, data, bytes, hipMemcpyHostToDevice, streamHd);
        if (err != 0)
        {
            printf("[cudaError] cudaMemcpyAsync返回0x%x\n", err);
            exit(1);
        }
        err = hipMemcpyAsync(list[cur].pDevResult, list[cur].pHostResult, bytes, hipMemcpyHostToDevice, streamHd);
        if (err != 0)
        {
            printf("[cudaError] cudaMemcpyAsync返回0x%x\n", err);
            exit(1);
        }
        hipStreamSynchronize(streamHd);
        list[cur].id = i;
        flag[1] = NEXT_TASK(cur);
        free(data);
        printf("[cpu] 在%d处插入任务%d\n", cur, i);
    }
    return NULL;
}

/*设备端消费者*/
__global__ void gpuConsumer(struct Task *devList, int *devFlag, int *devFinTaksNb)
{
    int threadId = threadIdx.x;
    while (devFinTaksNb[0] != TASK_NB)
    {
        __syncthreads();
        bool temp = false;
        while (devFlag[0] == devFlag[1])
        {
            if (threadId == 0)
            {
                if (temp == false)
                {
                    printf("[gpu] 队列是空的\n");
                    temp = true;
                }
            }
        }
        int cur = devFlag[0];
        if (threadId < devList[cur].nb)
        {
            int task = devList[cur].pData[threadId];
            devList[cur].pDevResult[threadId] = pow(task, 2) - task;
        }
        __syncthreads();
        if (threadId == 0)
        {
            printf("[gpu] %d处的任务%d处理完成\n", cur, devList[cur].id);
            devList[cur].isSave = true;
            while (devList[cur].isSave == true)
            {
            }
        }
    }
}

/*保存结果到文件*/
void *cpuSaver(void *argc)
{
    while (finTaksNb[0] != TASK_NB)
    {
        int cur = flag[0];
        if (list[cur].nb > 0)
        {
            while (list[cur].isSave == false)
            {
            }
            int bytes = sizeof(int) * list[cur].nb;
            err = hipMemcpyAsync(list[cur].pHostResult, list[cur].pDevResult, bytes, hipMemcpyDeviceToHost, streamDh);
            if (err != 0)
            {
                printf("[cudaError] cudaMemcpyAsync返回0x%x\n", err);
                exit(1);
            }
            hipStreamSynchronize(streamDh);
            FILE *fp = fopen("./result.txt", "a+");
            fprintf(fp, "%d\t", list[cur].id);
            for (int i = 0; i < list[cur].nb; i++)
            {
                fprintf(fp, "%d", list[cur].pHostResult[i]);
                if (i < list[cur].nb - 1)
                {
                    fprintf(fp, " ");
                }
            }
            fprintf(fp, "\n");
            fclose(fp);
            printf("[cpu] %d处的任务%d结果已经保存\n", cur, list[cur].id);
            list[cur].isSave = false;
            flag[0] = NEXT_TASK(cur);
            (finTaksNb[0])++;
        }
    }
    return NULL;
}

/*初始化*/
void init()
{
    remove("./result.txt");
    int listBytes = LIST_SIZE * sizeof(struct Task);
    int flagBytes = 2 * sizeof(int);

    hipHostMalloc((void **)&list, listBytes, hipHostMallocMapped, hipHostMallocDefault);
    hipHostMalloc((void **)&flag, flagBytes, hipHostMallocMapped, hipHostMallocDefault);
    hipHostMalloc((void **)&finTaksNb, sizeof(int), hipHostMallocMapped, hipHostMallocDefault);
    memset(flag, 0, flagBytes);
    memset(finTaksNb, 0, sizeof(int));

    for (int i = 0; i < LIST_SIZE; i++)
    {
        err = hipMalloc((void **)&(list[i].pData), sizeof(int) * WARP_SIZE);
        if (err != 0)
        {
            printf("[cudaError] cudaMalloc返回0x%x\n", err);
            exit(1);
        }
        err = hipMalloc((void **)&(list[i].pDevResult), sizeof(int) * WARP_SIZE);
        if (err != 0)
        {
            printf("[cudaError] cudaMalloc返回0x%x\n", err);
            exit(1);
        }
        list[i].pHostResult = (int *)malloc(sizeof(int) * WARP_SIZE);
        memset(list[i].pHostResult, 0, sizeof(int) * WARP_SIZE);
    }

    hipStreamCreate(&streamHd);
    hipStreamCreate(&streamDh);
    hipStreamCreate(&streamKernel);

    hipHostGetDevicePointer<struct Task>(&devList, (void *)list, 0);
    hipHostGetDevicePointer<int>(&devFlag, (void *)flag, 0);
    hipHostGetDevicePointer<int>(&devFinTaksNb, (void *)finTaksNb, 0);
}

/*清理*/
void free()
{
    hipStreamDestroy(streamHd);
    hipStreamDestroy(streamDh);
    hipStreamDestroy(streamKernel);

    for (int i = 0; i < LIST_SIZE; i++)
    {
        hipFree(list[i].pData);
        hipFree(list[i].pDevResult);
        free(list[i].pHostResult);
    }

    hipHostFree(list);
    hipHostFree(flag);
    hipHostFree(finTaksNb);
}

int main()
{
    init();

    pthread_t cpu_pro, cpu_sav;
    pthread_create(&cpu_sav, NULL, cpuSaver, NULL);
    gpuConsumer<<<1, WARP_SIZE, 0, streamKernel>>>(devList, devFlag, devFinTaksNb);
    pthread_create(&cpu_pro, NULL, cpuProducer, NULL);

    pthread_join(cpu_pro, NULL);
    printf("cpuProducer已经退出\n");
    hipDeviceSynchronize();
    printf("gpuConsumer已经退出\n");
    pthread_join(cpu_sav, NULL);
    printf("cpuSaver已经退出\n");

    free();
    return 0;
}

/*
*vscode的工作目录必须为cuda*
rm -rf cpu_gpu_syn cpu_gpu_syn.o result.txt
/usr/local/cuda/bin/nvcc -ccbin g++ -I /usr/local/cuda/include -I /usr/local/cuda/samples/common/inc -m64 -g -G -gencode arch=compute_75,code=sm_75 -gencode arch=compute_75,code=compute_75 -o cpu_gpu_syn.o -c cpu_gpu_syn.cu -dc
/usr/local/cuda/bin/nvcc -ccbin g++ -m64 -g -G -gencode arch=compute_75,code=sm_75 -gencode arch=compute_75,code=compute_75 -o cpu_gpu_syn cpu_gpu_syn.o -L /usr/local/cuda/lib64 -L /usr/local/cuda/samples/common/lib

./cpu_gpu_syn

cuda-gdb
file cpu_gpu_syn
r
q

rm -rf cpu_gpu_syn cpu_gpu_syn.o result.txt
*/