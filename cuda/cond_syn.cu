#include <stdio.h>
#include <hip/hip_runtime.h>
#include <bits/stdint-uintn.h>
#include <unistd.h>

#define THREADS_PER_GROUP 64
#define GROUP_NB 2

static char c;
static __device__ char devC;
static __device__ const int arrayNb = 2;

__global__ void cond_syn()
{
	/*该线程的ID*/
	int threadId = threadIdx.x;

	/*该线程所在组的ID*/
	int groupId = threadId / THREADS_PER_GROUP;

	/*该线程的相对ID*/
	int _threadId = threadId % THREADS_PER_GROUP;

	/*组的状态变量*/
	__shared__ int group[GROUP_NB];

	/*该组中已经完成任务的线程数量*/
	group[groupId] = THREADS_PER_GROUP;

	char array[arrayNb];
	array[0] = '-';
	array[1] = '*';

	for (int i = 0; i < 3; i++)
	{
		/*执行该线程的相关任务*/
		printf("%c%c groupId-%d-threadId-%d执行任务%d\n", array[0], array[1], groupId, threadId, i);

		/*组中最快的线程初始化组的状态变量为0*/
		atomicCAS((group + groupId), THREADS_PER_GROUP, 0);

		/*组中线程完成任务后更新组的状态变量*/
		int temp = atomicAdd((group + groupId), uint32_t(1));

		/*等待组中所有线程全部完成任务*/
		if (temp != THREADS_PER_GROUP - 1)
		{
			while (group[groupId] != THREADS_PER_GROUP)
			{
			}
		}

		if (_threadId == 0)
		{
			printf("%c groupId-%d完成任务%d\n", devC, groupId, i);
		}
	}
}

int main()
{
	char fileName[15] = "cond_syn.log";
	remove(fileName);
	int stdDup = dup(1);
	FILE *outLog = fopen(fileName, "a");
	dup2(fileno(outLog), 1);

	c = '$';
	hipMemcpyToSymbol(HIP_SYMBOL(devC), &c, sizeof(c));

	cond_syn<<<1, GROUP_NB * THREADS_PER_GROUP>>>();
	/*如果不加这句话main函数将不等cond_syn执行直接结束*/
	hipDeviceSynchronize();

	fflush(stdout);
	fclose(outLog);
	dup2(stdDup, 1);
	close(stdDup);

	return 0;
}
/*
*vscode的工作目录必须为cuda*

rm -rf cond_syn cond_syn.o
/usr/local/cuda/bin/nvcc -ccbin g++ -I /usr/local/cuda/include -I /usr/local/cuda/samples/common/inc -m64 -g -G -gencode arch=compute_75,code=sm_75 -gencode arch=compute_75,code=compute_75 -o cond_syn.o -c cond_syn.cu
/usr/local/cuda/bin/nvcc -ccbin g++ -m64 -g -G -gencode arch=compute_75,code=sm_75 -gencode arch=compute_75,code=compute_75 -o cond_syn cond_syn.o -L /usr/local/cuda/lib64 -L /usr/local/cuda/samples/common/lib

./cond_syn

cuda-gdb
file cond_syn
r
q

rm -rf cond_syn cond_syn.o cond_syn.log
*/