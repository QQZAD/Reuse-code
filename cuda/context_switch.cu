#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

/*
【warp上下文切换】微观
在CPU上，上下文切换是由内核中的一个名为“调度器”的函数在软件中完成的。
调度器是普通代码，是处理器必须运行的机器指令序列，而运行调度器所花费的时间是没有用于“有用”工作的时间。
一旦线程块在SM上启动，它的所有warp都将驻留，直到它们全部退出内核。GPU没有传统意义上的上下文切换。
SM更有可能从不同的warp而不是从相同的warp在一行中发出两条指令，如果不这样做，将使SM暴露于依赖暂停。

另一方面，GPU在硬件中进行上下文切换，而不需要调度器，而且它足够快。
当一个warp遇到"pipeline stall"时，另一个warp可以利用pipeline阶段，否则这些阶段将是空闲的。
这被称为“延迟隐藏”——一个warp的延迟被其他warp的进度所隐藏。
GPU使用上下文切换来隐藏延迟以获得更大的吞吐量。

【任务上下文切换】宏观
不同的kernel函数共享GPU上同一个SM，针对不同应用场景的三种抢占策略
1.把一个SM上正在运行的thread block(TB)的上下文保存到内存，启动一个新的kernel函数抢占当前SM。
其切换开销对吞吐量影响！中！，对延迟影响！中！。
2.等待一个SM上正在运行的kernel函数的所有TB结束，启动一个新的kernel函数抢占当前SM。
其切换开销对吞吐量影响！小！，对延迟影响！大！。
3.对于具有幂等性的kernel函数，即使强制结束当前正在运行的TB，重启后也不会对kernel函数的结果产生影响，不需要保存任何上下文信息。
其切换开销对吞吐量影响！大！（当抢占发生在任务即将结束时），对延迟影响！很小！。

幂等性：在编程中一个幂等操作的特点是其任意多次执行所产生的影响均与一次执行的影响相同。
使用相同参数重复执行能获得相同结果。不会影响系统状态，也不用担心重复执行会对系统造成改变。
*/

__global__ void contextSwitch()
{
    int threadId = threadIdx.x;
    while (hostFinTaksNb[0] != TASK_NB)
    {
        __syncthreads();
        bool temp = false;
        while (hostFlag[0] == hostFlag[1])
        {
            if (threadId == 0)
            {
                if (temp == false)
                {
                    printf("[gpu] 队列是空的\n");
                    temp = true;
                }
            }
        }
        int cur = hostFlag[0];
        int task = hostList[cur].pData[threadId];
        hostList[cur].pResult[threadId] = pow(task, 2) - task;
        __syncthreads();
        if (threadId == 0)
        {
            printf("[gpu] %d处的任务%d处理完成\n", cur, hostList[cur].id);
            hostList[cur].isSave = true;
            while (hostList[cur].isSave == true)
            {
            }
        }
    }
}

int main()
{
    contextSwitch<<<1, WARP_SIZE, 0, streamKernel>>>();
    hipDeviceSynchronize();
    return 0;
}
/*
*vscode的工作目录必须为cuda*

rm -rf context_switch context_switch.o
/usr/local/cuda/bin/nvcc -ccbin g++ -I /usr/local/cuda/include -I /usr/local/cuda/samples/common/inc -m64 -g -G -gencode arch=compute_75,code=sm_75 -gencode arch=compute_75,code=compute_75 -o context_switch.o -c context_switch.cu
/usr/local/cuda/bin/nvcc -ccbin g++ -m64 -g -G -gencode arch=compute_75,code=sm_75 -gencode arch=compute_75,code=compute_75 -o context_switch context_switch.o -L /usr/local/cuda/lib64 -L /usr/local/cuda/samples/common/lib

./context_switch

cuda-gdb
file context_switch
r
q

rm -rf context_switch context_switch.o
*/