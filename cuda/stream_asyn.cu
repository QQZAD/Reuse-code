#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#define nStreams 4
static hipEvent_t startEvent, stopEvent;
static hipStream_t stream[nStreams];

__global__ void kernel(float *a, int offset)
{
    int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
    float x = (float)i;
    float s = sinf(x);
    float c = cosf(x);
    a[i] += sqrtf(s * s + c * c);
}

void sequential(float *a, float *d_a, int bytes, int blockSize, int n)
{
    float ms;
    memset(a, 0, bytes);
    hipEventRecord(startEvent, 0);

    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    kernel<<<n / blockSize, blockSize>>>(d_a, 0);
    hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    printf("sequential数据传输和执行的总时间：%fms\n", ms);
}

void asynchronous1(float *a, float *d_a, int bytes, int blockSize, int streamSize, int streamBytes)
{
    float ms;
    memset(a, 0, bytes);
    hipEventRecord(startEvent, 0);

    for (int i = 0; i < nStreams; i++)
    {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_a[offset], &a[offset], streamBytes, hipMemcpyHostToDevice, stream[i]);
        kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
        hipMemcpyAsync(&a[offset], &d_a[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]);
    }

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    printf("asynchronous1数据传输和执行的总时间：%fms\n", ms);
}

void asynchronous2(float *a, float *d_a, int bytes, int blockSize, int streamSize, int streamBytes)
{
    float ms;
    memset(a, 0, bytes);
    hipEventRecord(startEvent, 0);

    for (int i = 0; i < nStreams; i++)
    {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_a[offset], &a[offset], streamBytes, hipMemcpyHostToDevice, stream[i]);
    }
    for (int i = 0; i < nStreams; i++)
    {
        int offset = i * streamSize;
        kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
    }
    for (int i = 0; i < nStreams; i++)
    {
        int offset = i * streamSize;
        hipMemcpyAsync(&a[offset], &d_a[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]);
    }

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    printf("asynchronous2数据传输和执行的总时间：%fms\n", ms);
}

int main()
{
    int blockSize = 256;
    int n = 4 * 1024 * blockSize * nStreams;
    int streamSize = n / nStreams;
    int streamBytes = streamSize * sizeof(float);
    int bytes = n * sizeof(float);

    int devId = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devId);
    printf("是否支持执行与数据传输重叠：%d\n", prop.deviceOverlap);
    printf("异步引擎的数量：%d\n", prop.asyncEngineCount);
    hipSetDevice(devId);

    /*分配设备内存和pinned主机内存*/
    float *a, *d_a;
    hipHostMalloc((void **)&a, bytes);
    hipMalloc((void **)&d_a, bytes);

    /*创建开始事件和结束事件*/
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    /*创建Non-default流*/
    for (int i = 0; i < nStreams; i++)
    {
        hipStreamCreate(&stream[i]);
    }

    sequential(a, d_a, bytes, blockSize, n);
    asynchronous1(a, d_a, bytes, blockSize, streamSize, streamBytes);
    asynchronous2(a, d_a, bytes, blockSize, streamSize, streamBytes);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    for (int i = 0; i < nStreams; ++i)
    {
        hipStreamDestroy(stream[i]);
    }
    hipFree(d_a);
    hipHostFree(a);

    return 0;
}
/*
*vscode的工作目录必须为cuda*

rm -rf stream_asyn stream_asyn.o
/usr/local/cuda/bin/nvcc -ccbin g++ -I /usr/local/cuda/include -I /usr/local/cuda/samples/common/inc -m64 -g -G -gencode arch=compute_75,code=sm_75 -gencode arch=compute_75,code=compute_75 -o stream_asyn.o -c stream_asyn.cu
/usr/local/cuda/bin/nvcc -ccbin g++ -m64 -g -G -gencode arch=compute_75,code=sm_75 -gencode arch=compute_75,code=compute_75 -o stream_asyn stream_asyn.o -L /usr/local/cuda/lib64 -L /usr/local/cuda/samples/common/lib

./stream_asyn

cuda-gdb
file stream_asyn
r
q

rm -rf stream_asyn stream_asyn.o
*/